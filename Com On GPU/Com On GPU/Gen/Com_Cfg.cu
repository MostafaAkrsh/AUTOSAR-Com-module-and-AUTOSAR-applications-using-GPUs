/*******************************************************************************************************************************
FileName:                                               Com_Cfg.c
AUTOSAR Version:                                          4.2.2
******************************************************************************************************************************/
/******************************************************************************************************************************
 **                                                     Includes                                                             **
 ******************************************************************************************************************************/
#pragma once

#include "Com_Cfg.cuh"
#include "../Inc/Com_Types.cuh"

// maybe they should be in the .cuh
#include "hip/hip_runtime.h"
#include ""

/*****************************************************************************************************************************
 **                                         Post-Build Configuration variables values                                       **
 *****************************************************************************************************************************/

/* ComSignal Buffers */
__device__ uint8 image_signal_buffer[WIDTH * HEIGHT * CHANNEL_NUM];

/* ComGroupSignal Buffers */

/* Com IPdu Buffers */
__device__ uint8 image_IPDU_buffer[WIDTH * HEIGHT * CHANNEL_NUM + 1];

// fucntions removed

// Test test{ {1, 2, 3} };
// const void* my_symbol = image_signal_buffer;

// Com is now an array of one element so it can be copied between host and device
// check pointers


__device__ Com_Type Com_GPU[1] =
{
{
        //.ComConfig =
        {
        // .ComSignal =
        {
            { //
             12, // .ComBitPosition =  // try 25
             WIDTH * HEIGHT * CHANNEL_NUM * 8,	// .ComBitSize =
             0, // .ComHandleId =
             NULL, // .ComNotification =  
             LITTLE_ENDIAN, // .ComSignalEndianness = 
             WIDTH * HEIGHT * CHANNEL_NUM , // .ComSignalLength = 
             _UINT8,// .ComSignalType = 
             TRIGGERED, // .ComTransferProperty =  
             WIDTH * HEIGHT * CHANNEL_NUM * 8, // .ComUpdateBitPosition = 
             COM_SIGNAL_INIT_VALUE, // .ComSignalInitValue = 
             image_signal_buffer, // .ComBufferRef = 
             0 // .ComIpduHandler =
            }
        },

    //.ComIPdu =
    {
        {
           Receive, // .ComIPduDirection = 
           0, // .ComIPduHandleId = 
           IMMEDIATE, // .ComIPduSignalProcessing = 
           NORMAL, // .ComIPduType = 
           // .ComIPduGroupRef =
            {
                NULL
            },
    // .ComIPduSignalGroupRef =
     {
         NULL
     },
    // .ComIPduSignalRef =
     {
         // &Com.ComConfig.ComSignal[0], // causes circular dependency
         NULL
     },
    // .ComTxIPdu =
    {
        0.5, //.ComMinimumDelayTime = 
        Confirmation, // .ComTxIPduClearUpdateBit = 
         255, // .ComTxIPduUnusedAreasDefault =
        // .ComTxModeFalse =
        {
        // .ComTxMode =
        {
            PERIODIC, // .ComTxModeMode = 
            2, // .ComTxModeNumberOfRepetitions = 
            2, // .ComTxModeRepetitionPeriod =
            2 // .ComTxModeTimePeriod = 
        }
    }
},
image_IPDU_buffer, // .ComBufferRef = 
WIDTH * HEIGHT * CHANNEL_NUM + 1 // .ComIPduLength = 
}
}
}
}
};
