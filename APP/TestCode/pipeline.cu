#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define CHANNEL_NUM 3
#define NUM_REPS 5
#define TILE_DIM 32

__global__ void rgb2gray(uint8_t* out_img, uint8_t* in_img,
    int width, int height, int channels);
__global__ void doub_thresh(uint8_t* out_img, uint8_t* in_img,
    uint8_t lower_limit, uint8_t upper_limit,
    int width, int height);
__global__ void thresh2lanes(uint8_t* red_roads_img, uint8_t* edges_img,
                        int width, int height, int channels);
__global__ void copy(uint8_t* out_img, uint8_t* in_img,
    int width, int height, int channels);


int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int width, height, channels;
    uint8_t* rgb_in = stbi_load("image.jpg", &width, &height, &channels, 0);
    size_t rgb_size = width * height * CHANNEL_NUM * sizeof(uint8_t);
    printf("%d %d %d\n", width, height, channels);

    uint8_t * red_roads_out;
    red_roads_out = (uint8_t*)malloc(width * height * CHANNEL_NUM);
    size_t red_roads_size = rgb_size;

    uint8_t* gray_out;
    gray_out = (uint8_t*)malloc(width * height * 1);
    size_t gray_size = width * height * 1 * sizeof(uint8_t);

    uint8_t* edges_out;
    edges_out = (uint8_t*)malloc(width * height * 1);
    size_t edges_size = gray_size;

    uint8_t* d_rgb_in; uint8_t* d_gray_out; uint8_t* d_edges_out; uint8_t* d_red_roads_out;
    hipMalloc((void**)&d_rgb_in, rgb_size);
    hipMalloc((void**)&d_gray_out, gray_size);
    hipMalloc((void**)&d_edges_out, edges_size);
    hipMalloc((void**)&d_red_roads_out, red_roads_size);

    dim3 dimGrid(ceil(width / 32.0), ceil(height / 32.0), 1);
    dim3 dimBlock(32, 32, 1);

    hipEventRecord(start);
    for (int rep = 0; rep < NUM_REPS; rep++) {
        hipMemcpy(d_rgb_in, rgb_in, rgb_size, hipMemcpyHostToDevice);
        hipMemcpy(d_red_roads_out, d_rgb_in, red_roads_size, hipMemcpyDeviceToDevice);


        rgb2gray << <dimGrid, dimBlock >> > (d_gray_out, d_rgb_in, width, height, CHANNEL_NUM);

        doub_thresh << <dimGrid, dimBlock >> > (d_edges_out, d_gray_out, 180, 250, width, height);

        thresh2lanes << <dimGrid, dimBlock >> > (d_red_roads_out, d_edges_out, width, height, CHANNEL_NUM);


        // hipMemcpy(gray_out, d_gray_out, gray_size, hipMemcpyDeviceToHost);
        // hipMemcpy(edges_out, d_edges_out, gray_size, hipMemcpyDeviceToHost);
        hipMemcpy(red_roads_out, d_red_roads_out, red_roads_size, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms", milliseconds / NUM_REPS);

    // stbi_write_jpg("results/out.jpg", width, height, 1, gray_out, 100);
    // stbi_write_jpg("results/edges.jpg", width, height, 1, edges_out, 100);
    stbi_write_jpg("red_lanes.jpg", width, height, CHANNEL_NUM, red_roads_out, 100);
    
    stbi_image_free(rgb_in);
    free(gray_out);
    free(edges_out);
    free(red_roads_out);

    hipFree(d_rgb_in);
    hipFree(d_gray_out);
    hipFree(d_edges_out);
    hipFree(d_red_roads_out);

    return 0;
}

__global__ void copy(uint8_t* out_img, uint8_t* in_img,
    int width, int height, int channels) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    int base_index = row * width + col;

    if (row < height && col < width) {
        out_img[base_index * channels + 0] = in_img[base_index * channels + 0];
        out_img[base_index * channels + 1] = in_img[base_index * channels + 1];
        out_img[base_index * channels + 2] = in_img[base_index * channels + 2];
    }
}

__global__ void rgb2gray(uint8_t * out_img, uint8_t * in_img, 
                         int width, int height, int channels) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    int base_index = row * width + col;

    if (row < height && col < width) {
        uint8_t red = in_img[base_index * channels + 0];
        uint8_t green = in_img[base_index * channels + 1];
        uint8_t blue = in_img[base_index * channels + 2];
        uint8_t gray = 0.21f * red + 0.71f * green + 0.07 * blue;
        // uint8_t gray = (red + green +  blue) / 3;

        out_img[base_index] = gray;
    }
}

__global__ void doub_thresh(uint8_t* out_img, uint8_t* in_img, 
                            uint8_t lower_limit, uint8_t upper_limit, 
                            int width, int height) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    int base_index = row * width + col;

    if (row < height && col < width) {
        uint8_t pixel = in_img[base_index];
        if (pixel > lower_limit && pixel < upper_limit)
            out_img[base_index] = 240;
        else out_img[base_index] = 0;
    }
}


__global__ void thresh2lanes(uint8_t* red_roads_img, uint8_t* edges_img,
    int width, int height, int channels) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    int base_index = row * width + col;
    if (row < height && col < width) {

        if (row > (height * 0.6) && edges_img[base_index] != 0) {
            // coalescing may be needed here because edges are vertical
            // but the input image need to be transposed from the beggining (read)

            red_roads_img[base_index * channels + 0] = 250; // red
            red_roads_img[base_index * channels + 1] = 0; // no green
            red_roads_img[base_index * channels + 2] = 0; // no blue
        }
    }
}
